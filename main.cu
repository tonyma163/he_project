#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstddef>
#include <fstream>
#include <iostream>
#include <mutex>
#include <random>
#include <vector>
#include "phantom.h"
#include "util.cuh"
#include <cstdlib>
#include <sys/stat.h>
#include <filesystem>

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

//Set to True to test the program on the IDE
bool IDE_MODE = true;

string input_folder;

//Argument
string text;

void setup_environment(int argc, char *argv[]);
static inline vector<double> read_values_from_file(const string& filename, double scale);
//static inline PhantomPlaintext read_plain_repeated_input(PhantomContext context, PhantomPlaintext plaintext, const string& filename, double scale);

int main(int argc, char *argv[]) {
    // ./app_name.cu "{input_text}"
    setup_environment(argc, argv);

    // Encryption parameters
    EncryptionParameters parms(scheme_type::ckks);
    //size_t poly_modulus_degree = 16384;
    size_t poly_modulus_degree = 32768; // 16384*2
    parms.set_poly_modulus_degree(poly_modulus_degree);

    // Coefficient modulus
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60}
    ));

    // Context
    PhantomContext context(parms);

    // Keys
    // secret & public keys
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    // relinearization keys for multiplication
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    // galois keys for rotation
    PhantomGaloisKey galois_keys = secret_key.create_galois_keys(context);

    // Encoder
    PhantomCKKSEncoder encoder(context);

    // Scale parameter
    double scale = pow(2.0, 40);

    // Load embeddings
    //vector<double> input_embeddings;

    // no. of files in specified directory
    int inputs_count = 0;
    //filesystem::path path { "../python/tmp_embeddings" };
    /*
    for (__attribute__((unused)) auto& p : filesystem::directory_iterator("../python/tmp_embeddings")) {
        ++inputs_count;
    }
    */
    for (const auto& entry : filesystem::directory_iterator("../python/tmp_embeddings")) {
        ++inputs_count;
    }

    vector<PhantomCiphertext> inputs;
    for (int i=0; i<inputs_count; i++ ) {
        string filename = "../python/tmp_embeddings/input_"+to_string(i)+".txt";
        vector<double> input_embeddings = read_values_from_file(filename, scale);
        vector<double> repeated;

        // check
        if (input_embeddings.size() < 128) {
            cerr << "Not enough embeddings in file: " << endl;
            continue;
        }        

        for (int j=0; j<128; j++) { // 128 bert-tiny hidden layer
            for (int k=0; k<128; k++) {
                repeated.push_back(input_embeddings[j]);
            }
        }

        //cout << "size: " << repeated.size() << endl;

        // encrypt input embeddings
        //cout << "repeated" << repeated[0]; // test
        PhantomPlaintext plaintext;
        encoder.encode(context, repeated, scale, plaintext);
        
        PhantomCiphertext ciphertext;
        public_key.encrypt_asymmetric(context, plaintext, ciphertext);

        inputs.push_back(ciphertext);
    }

    // Encoder1
    vector<double> query_weight_vec = read_values_from_file("../weights-sst2/layer0_attself_query_weight.txt", scale);
    PhantomPlaintext query_w_pt;
    encoder.encode(context, query_weight_vec, scale, query_w_pt);

    /*
    vector<double> query_bias_vec = read_values_from_file("../weights-sst2/layer0_attself_query_weight.txt", scale);
    
    for (int j=0; j<128; j++) { // 128 bert-tiny hidden layer
        for (int k=0; k<128; k++) {
            repeated.push_back(input_embeddings[j]);
        }
    }

    PhantomPlaintext query_w_pt;
    encoder.encode(context, query_weight_vec, scale, query_w_pt);
    */

    return 0;
}

void setup_environment(int argc, char *argv[]) {
    string command;

    cout << "argc: " << argc;

    if (IDE_MODE) {
        //Removing any previous embedding
        filesystem::remove_all("../python/tmp_embeddings");

        input_folder = "../python/tmp_embeddings/";

        //text = "This is a bad movie.";
        text = argv[1];
        command = "python3 ../python/ExtractEmbeddings.py \"" + text + "\"";

        system(command.c_str());

        return;
    }
}

static inline vector<double> read_values_from_file(const string& filename, double scale) {
    vector<double> values;
    
    ifstream file(filename);
    /*
    if (!file.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        return values;
    }
    */

    string row;
    while (getline(file, row)) {
        istringstream stream(row);
        string value;
        while (getline(stream, value, ',')) {
            try {
                double num = stod(value);
                values.push_back(num * scale);
            } catch (const invalid_argument& e) {
                cerr << "Cannot convert: " << value << endl;
            }
        }
    }
    file.close();
    return values;
}

/*
static inline PhantomPlaintext read_plain_repeated_input(PhantomContext context, PhantomPlaintext plaintext, const string& filename, double scale) {
    vector<double> input = read_values_from_file(filename);

    vector<double> repeated;
    // check
    if (input_embeddings.size() < 128) {
        cerr << "Not enough embeddings in file: " << endl;
        continue;
    }

    for (int j=0; j<128; j++) { // 128 bert-tiny hidden layer
        for (int k=0; k<128; k++) {
            repeated.push_back(input_embeddings[j]);
        }
    }

    encoder.encode(context, repeated, scale, plaintext);

    return plaintext;
}
*/