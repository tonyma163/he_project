#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstddef>
#include <fstream>
#include <iostream>
#include <mutex>
#include <random>
#include <vector>
#include "phantom.h"
#include "util.cuh"
#include <cstdlib>
#include <sys/stat.h>
#include <filesystem>

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

//Set to True to test the program on the IDE
bool IDE_MODE = true;

string input_folder;

//Argument
string text;

void setup_environment(int argc, char *argv[]);

int main(int argc, char *argv[]) {
    // ./app_name.cu "{input_text}"
    setup_environment(argc, argv);

    // Encryption parameters
    EncryptionParameters parms(scheme_type::ckks);
    size_t poly_modulus_degree = 16384;
    parms.set_poly_modulus_degree(poly_modulus_degree);

    // Coefficient modulus
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 60}
    ));

    // Context
    PhantomContext context(parms);

    // Keys
    // secret & public keys
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    // relinearization keys for multiplication
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    // galois keys for rotation
    PhantomGaloisKey galois_keys = secret_key.create_galois_keys(context);

    // Encoder
    PhantomCKKSEncoder encoder(context);

    // Scale parameter
    double scale = pow(2.0, 40);

    // Load embeddings
    vector<double> input_embeddings;

    // no. of files in specified directory
    int inputs_count = 0;
    filesystem::path path { "../python/tmp_embeddings" };
    for (__attribute__((unused)) auto& p : filesystem::directory_iterator(path)) {
        ++inputs_count;
    }

    for (int i=0; i<inputs_count; i++ ) {
        string path = "../python/tmp_embeddings/input_"+to_string(i)+".txt";

        ifstream file(path);

        string row;
        while (getline(file, row)) {
            istringstream stream(row);
            string value;
            while (getline(stream, value, ',')) {
                try {
                    double num = stod(value);
                    input_embeddings.push_back(num * scale);
                } catch (const invalid_argument e) {
                    cerr << "Cannot convert: " << value << endl;
                }
            }
        }
        file.close();

        vector<double> repeated;

        for (int j=0; j<128; j++) { // 128 bert-tiny hidden layer
            for (int i=0; i<128; i++) {
                repeated.push_back(input_embeddings[j]);
            }
        }

        int size = static_cast<int>(repeated.size());

        if (scale != 1) {
            for (int i=0; i<size; i++) {
                repeated[i] = repeated[i] * scale;
            }
        }

        // encrypt it
        cout << "repeated" << repeated[0]; // test
    }
    
    

    // Encoder1

    return 0;
}

void setup_environment(int argc, char *argv[]) {
    string command;

    cout << "argc: " << argc;

    if (IDE_MODE) {
        //Removing any previous embedding
        filesystem::remove_all("../python/tmp_embeddings");

        input_folder = "../python/tmp_embeddings/";

        //text = "This is a bad movie.";
        text = argv[1];
        command = "python3 ../python/ExtractEmbeddings.py \"" + text + "\"";

        system(command.c_str());

        return;
    }
}